#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/winograd_layer.hpp"
#include "caffe/util/winograd.hpp"

namespace caffe {

template <typename Dtype>
__global__ void winograd_input_im2col_gpu_kernel(
  const int n,
  const Dtype *data, Dtype *col_buff,
  int height, int width,
  int pad_h, int pad_w,
  int ntiles_h, int ntiles_w,
  int tile_h_in, int tile_w_in,
  int tile_h_out, int tile_w_out)
{
  CUDA_KERNEL_LOOP(index, n) {
    const int x = index%tile_w_in;
    const int y = index/tile_w_in%tile_h_in;
    const int tile_w = index/tile_w_in/tile_h_in%ntiles_w;
    const int tile_h = index/tile_w_in/tile_h_in/ntiles_w%ntiles_h;
    const int c = index/tile_w_in/tile_h_in/ntiles_w/ntiles_h;

    int in_y = tile_h*tile_h_out + y - pad_h;
    int in_x = tile_w*tile_w_out + x - pad_w;

    if (in_y < 0 || in_x < 0 || in_y >= height || in_x >= width) {
      col_buff[(((c*ntiles_h + tile_h)*ntiles_w + tile_w)*tile_h_in + y)*tile_w_in + x] = 0;
    }
    else {
      col_buff[(((c*ntiles_h + tile_h)*ntiles_w + tile_w)*tile_h_in + y)*tile_w_in + x] = data[(c*height + in_y)*width + in_x];
    }
  }
}

template <typename Dtype>
__global__ void winograd_output_col2im_gpu_kernel(
  const int n,
  const Dtype *col_buff, Dtype *data,
  int output_h, int output_w,
  int ntiles_h, int ntiles_w,
  int tile_h_out, int tile_w_out)
{
  CUDA_KERNEL_LOOP(index, n) {
    const int x = index%tile_w_out;
    const int y = index/tile_w_out%tile_h_out;
    const int tile_w = index/tile_w_out/tile_h_out%ntiles_w;
    const int tile_h = index/tile_w_out/tile_h_out/ntiles_w%ntiles_h;
    const int c = index/tile_w_out/tile_h_out/ntiles_w/ntiles_h;

    int out_y = tile_h*tile_h_out + y;
    int out_x = tile_w*tile_w_out + x;

    if (out_y < output_h && out_x < output_w) {
      data[(c*output_h + out_y)*output_w + out_x] =
          col_buff[(((c*ntiles_h + tile_h)*ntiles_w + tile_w)*tile_h_out + y)*tile_w_out + x];
    }
  }
}

template <typename Dtype>
__global__ void winograd_output_im2col_gpu_kernel(
  const int n,
  const Dtype *data, Dtype *col_buff,
  int output_h, int output_w,
  int ntiles_h, int ntiles_w,
  int tile_h_out, int tile_w_out)
{
  CUDA_KERNEL_LOOP(index, n) {
    const int x = index%tile_w_out;
    const int y = index/tile_w_out%tile_h_out;
    const int tile_w = index/tile_w_out/tile_h_out%ntiles_w;
    const int tile_h = index/tile_w_out/tile_h_out/ntiles_w%ntiles_h;
    const int c = index/tile_w_out/tile_h_out/ntiles_w/ntiles_h;

    int out_y = tile_h*tile_h_out + y;
    int out_x = tile_w*tile_w_out + x;

    if (out_y < 0 || out_x < 0 || out_y >= output_h || out_x >= output_w) {
      col_buff[(((c*ntiles_h + tile_h)*ntiles_w + tile_w)*tile_h_out + y)*tile_w_out + x] = 0;
    }
    else {
      col_buff[(((c*ntiles_h + tile_h)*ntiles_w + tile_w)*tile_h_out + y)*tile_w_out + x] =
          data[(c*output_h + out_y)*output_w + out_x];
    }
  }
}

template <typename Dtype>
__global__ void winograd_input_col2im_gpu_kernel(
  const int n,
  const Dtype *col_buff, Dtype *data,
  int height, int width,
  int pad_h, int pad_w,
  int ntiles_h, int ntiles_w,
  int tile_h_in, int tile_w_in,
  int tile_h_out, int tile_w_out)
{
  CUDA_KERNEL_LOOP(index, n) {
    const int x = index%tile_w_in;
    const int y = index/tile_w_in%tile_h_in;
    const int tile_w = index/tile_w_in/tile_h_in%ntiles_w;
    const int tile_h = index/tile_w_in/tile_h_in/ntiles_w%ntiles_h;
    const int c = index/tile_w_in/tile_h_in/ntiles_w/ntiles_h;

    int in_y = tile_h*tile_h_out + y - pad_h;
    int in_x = tile_w*tile_w_out + x - pad_w;

    if (in_y >= 0 && in_x >= 0 && in_y < height && in_x < width) {
      data[(c*height + in_y)*width + in_x] +=
          col_buff[(((c*ntiles_h + tile_h)*ntiles_w + tile_w)*tile_h_in + y)*tile_w_in + x];
    }
  }
}

template <>
void WinogradLayer<double>::Forward_gpu(const vector<Blob<double>*>& bottom,
      const vector<Blob<double>*>& top) {
  NOT_IMPLEMENTED;
}

//#define PROFILE_WINOGRAD

template <>
void WinogradLayer<float>::Forward_gpu(const vector<Blob<float>*>& bottom,
      const vector<Blob<float>*>& top) {

  int kernel_h = this->kernel_shape_.cpu_data()[0], kernel_w = this->kernel_shape_.cpu_data()[1];

  WinogradAKronA<float> *AKronA = WinogradAKronA<float>::getInstance(kernel_h);
  WinogradBKronB<float> *BKronB = WinogradBKronB<float>::getInstance(kernel_h);
  WinogradGKronG<float> *GKronG = WinogradGKronG<float>::getInstance(kernel_h);

  const float* weight = this->blobs_[0]->gpu_data();

#ifdef PROFILE_WINOGRAD
  CPUTimer timer;
#endif

  for (int i = 0; i < bottom.size(); ++i) {
    const float* bottom_data = bottom[i]->gpu_data();
    float* top_data = top[i]->mutable_gpu_data();

    int M = this->conv_in_channels_*ntiles_h_*ntiles_w_;
    int num_kernels = this->num_*this->conv_in_channels_*ntiles_h_*ntiles_w_*tile_h_in_*tile_w_in_;
    int height = this->conv_input_shape_.cpu_data()[1], width = this->conv_input_shape_.cpu_data()[2];
    int pad_h = this->pad_.cpu_data()[0], pad_w = this->pad_.cpu_data()[1];

#ifdef PROFILE_WINOGRAD
    timer.Start();
#endif
    winograd_input_im2col_gpu_kernel<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                              CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, bottom_data, temp2_.mutable_gpu_data(),
      height, width,
      pad_h, pad_w,
      ntiles_h_, ntiles_w_,
      tile_h_in_, tile_w_in_,
      tile_h_out_, tile_w_out_);
    CUDA_POST_KERNEL_CHECK;
#ifdef PROFILE_WINOGRAD
    LOG(INFO) << "winograd_input_im2col takes " << timer.MicroSeconds()/1e6;
#endif

    // Transform input to Winograd domain
#ifdef PROFILE_WINOGRAD
    timer.Start();
#endif
    caffe_gpu_gemm<float>(CblasTrans, CblasTrans,
      tile_h_in_*tile_w_in_, this->num_*M, tile_h_in_*tile_w_in_,
      (float)1, BKronB->get()->gpu_data(), temp2_.mutable_gpu_data(),
      (float)0, temp1_.mutable_gpu_data());
      // temp1_ has (tile_h_in*tile_w_in) x num_ x (conv_in_channels) x (ntiles_h*ntiles_w) dimension
#ifdef PROFILE_WINOGRAD
    LOG(INFO) << "Transformation of bottom takes " << timer.MicroSeconds()/1e6;
#endif

#ifdef PROFILE_WINOGRAD
    timer.Start();
#endif
    // Convolution in Winograd domain
    {
      float alpha = 1, beta = 0;

      int M = this->conv_out_channels_/this->group_;
      int N = ntiles_h_*ntiles_w_;
      int K = this->conv_in_channels_/this->group_;

      if (!weight_ptrs_initialized_) {
        float **weight_ptrs = (float **)weight_ptrs_->mutable_cpu_data();
        for (int n = 0; n < this->num_; ++n) {
          for (int j = 0; j < tile_h_in_*tile_w_in_*this->group_; ++j) {
            weight_ptrs[n*tile_h_in_*tile_w_in_*this->group_ + j] = 
              this->blobs_[0]->mutable_gpu_data() +
              j*(this->conv_out_channels_/this->group_)*(this->conv_in_channels_/this->group_);
          }
        }
        weight_ptrs_initialized_ = true;
      }

      // TODO: instead of tile_h_in_ x tile_w_in_ x num_ instances of 
      // N x C x (ntiles_h_*ntiles_w_) GEMMs,
      // use tile_h_in_ x tile_w_in_ instances of
      // N x C x (num_*ntiles_h_*ntiles_w_) GEMMs
      CUBLAS_CHECK(hipblasSgemmBatched(
        Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        (const float **)in_activation_ptrs_->gpu_data(), N,
        (const float **)weight_ptrs_->gpu_data(), K,
        &beta,
        (float **)out_activation_ptrs_->mutable_gpu_data(), N,
        in_activation_ptrs_->count()));
    }
      // col_buff has num_ x (tile_h_in*tile_w_in) x (conv_out_channels) x (ntiles_h*ntiles_w)
#ifdef PROFILE_WINOGRAD
    LOG(INFO) << "Convolution takes " << timer.MicroSeconds()/1e6;
#endif

    // Transform back to time domain
#ifdef PROFILE_WINOGRAD
    timer.Start();
#endif
    caffe_gpu_gemm<float>(CblasTrans, CblasNoTrans,
        this->num_*this->conv_out_channels_*ntiles_h_*ntiles_w_, tile_h_out_*tile_w_out_, tile_h_in_*tile_w_in_,
        (float)1, temp2_.gpu_data(), AKronA->get()->gpu_data(),
        (float)0, temp1_.mutable_gpu_data());
#ifdef PROFILE_WINOGRAD
    LOG(INFO) << "Inverse transformation of top takes " << timer.MicroSeconds()/1e6;
#endif

#ifdef PROFILE_WINOGRAD
    timer.Start();
#endif
    num_kernels = this->num_*this->conv_out_channels_*ntiles_h_*ntiles_w_*tile_h_out_*tile_w_out_;
    const int output_h = this->output_shape_[0], output_w = this->output_shape_[1];
    winograd_output_col2im_gpu_kernel<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                               CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels,
      temp1_.gpu_data(), top_data,
      output_h, output_w,
      ntiles_h_, ntiles_w_,
      tile_h_out_, tile_w_out_); 
    CUDA_POST_KERNEL_CHECK;
#ifdef PROFILE_WINOGRAD
    LOG(INFO) << "winograd_output_col2im takes " << timer.MicroSeconds()/1e6;
#endif

    for (int n = 0; n < this->num_; ++n) { // JSP: this->num_ is batch size
      if (this->bias_term_) {
        const float* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
}

template <>
void WinogradLayer<double>::Backward_gpu(const vector<Blob<double>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<double>*>& bottom) {
  NOT_IMPLEMENTED;
}

template <>
void WinogradLayer<float>::Backward_gpu(const vector<Blob<float>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<float>*>& bottom) {

  int kernel_h = this->kernel_shape_.cpu_data()[0], kernel_w = this->kernel_shape_.cpu_data()[1];

  WinogradAKronA<float> *AKronA = WinogradAKronA<float>::getInstance(kernel_h);
  WinogradBKronB<float> *BKronB = WinogradBKronB<float>::getInstance(kernel_h);
  WinogradGKronG<float> *GKronG = WinogradGKronG<float>::getInstance(kernel_h);

  const float* weight = this->blobs_[0]->gpu_data();
  float* weight_diff = this->blobs_[0]->mutable_gpu_diff();

	/*const float *weight_cpu = this->blobs_[0]->cpu_data();
  fprintf(stderr, "weight_winograd\n");
  for (int j = 0; j < tile_h_in_*tile_w_in_; ++j) {
    for (int n = 0; n < this->conv_out_channels_; ++n) {
      for (int c = 0; c < this->conv_in_channels_; ++c) {
        fprintf(stderr, "%g ", weight_cpu[(j*this->conv_out_channels_ + n)*this->conv_in_channels_ + c]);
      }
    }
    fprintf(stderr, "\n");
  }*/

#ifdef PROFILE_WINOGRAD
  CPUTimer timer;
#endif

  for (int i = 0; i < top.size(); ++i) {
    const float* top_diff = top[i]->gpu_diff();
    const float* bottom_data = bottom[i]->gpu_data();
    float* bottom_diff = bottom[i]->mutable_gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      float* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      int M = this->conv_out_channels_*ntiles_h_*ntiles_w_;
      int num_kernels = this->num_*this->conv_out_channels_*ntiles_h_*ntiles_w_*tile_h_out_*tile_w_out_;
      const int output_h = this->output_shape_[0], output_w = this->output_shape_[1];
      const int height = this->conv_input_shape_.cpu_data()[1], width = this->conv_input_shape_.cpu_data()[2];
      const int pad_h = this->pad_.cpu_data()[0], pad_w = this->pad_.cpu_data()[1];

#ifdef PROFILE_WINOGRAD
      timer.Start();
#endif
      winograd_output_im2col_gpu_kernel<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                                 CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels,
        top_diff, temp1_.mutable_gpu_data(),
        output_h, output_w,
        ntiles_h_, ntiles_w_,
        tile_h_out_, tile_w_out_);
      CUDA_POST_KERNEL_CHECK;
#ifdef PROFILE_WINOGRAD
      LOG(INFO) << "winograd_output_im2col takes " << timer.MicroSeconds()/1e6;
#endif

      // Transform out_diff to Winograd domain
#ifdef PROFILE_WINOGRAD
      timer.Start();
#endif
      caffe_gpu_gemm<float>(CblasNoTrans, CblasTrans,
          tile_h_in_*tile_w_in_, this->num_*M, tile_h_out_*tile_w_out_,
          (float)1, AKronA->get()->gpu_data(), temp1_.mutable_gpu_data(),
          (float)0, temp2_.mutable_gpu_data());
      // temp2_ has (tile_h_in*tile_w_in) x num_ x (conv_out_channels) x (ntiles_h*ntiles_w) dimension
#ifdef PROFILE_WINOGRAD
      LOG(INFO) << "Transformation of top_diff takes " << timer.MicroSeconds()/1e6;
#endif

      // gradient w.r.t. weight. Note that we will accumulate diffs.
      if (this->param_propagate_down_[0]) {
#ifdef PROFILE_WINOGRAD
        timer.Start();
#endif
        int num_kernels = this->num_*this->conv_in_channels_*ntiles_h_*ntiles_w_*tile_h_in_*tile_w_in_;

        winograd_input_im2col_gpu_kernel<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                                  CAFFE_CUDA_NUM_THREADS>>>(
          num_kernels, bottom_data, this->col_buffer_.mutable_gpu_data(),
          height, width,
          pad_h, pad_w,
          ntiles_h_, ntiles_w_,
          tile_h_in_, tile_w_in_,
          tile_h_out_, tile_w_out_);
        CUDA_POST_KERNEL_CHECK;
#ifdef PROFILE_WINOGRAD
        LOG(INFO) << "winograd_input_im2col takes " << timer.MicroSeconds()/1e6;
#endif

        // Transform input to Winograd domain
#ifdef PROFILE_WINOGRAD
        timer.Start();
#endif
        caffe_gpu_gemm<float>(CblasTrans, CblasTrans,
            tile_h_in_*tile_w_in_, this->num_*this->conv_in_channels_*ntiles_h_*ntiles_w_, tile_h_in_*tile_w_in_,
            (float)1, BKronB->get()->gpu_data(), this->col_buffer_.mutable_gpu_data(),
            (float)0, temp1_.mutable_gpu_data());
        // temp1_ has (tile_h_in*tile_w_in) x num_ x (conv_in_channels) x (ntiles_h*ntiles_w) dimension
#ifdef PROFILE_WINOGRAD
        LOG(INFO) << "Transformation of bottom takes " << timer.MicroSeconds()/1e6;
#endif

        if (false/*n == 0*/) {
          const float *weight_diff_cpu = this->blobs_[0]->cpu_diff();
          fprintf(stderr, "weight_diff_winograd0[0]\n");
          for (int j = 0; j < tile_h_in_*tile_w_in_; ++j) {
            for (int n = 0; n < this->conv_out_channels_; ++n) {
              for (int c = 0; c < this->conv_in_channels_; ++c) {
                fprintf(stderr, "%g ", weight_diff_cpu[(j*this->conv_out_channels_ + n)*this->conv_in_channels_ + c]);
              }
            }
            fprintf(stderr, "\n");
          }
        }

#ifdef PROFILE_WINOGRAD
        timer.Start();
#endif

        if (!weight_diff_ptrs_initialized_) {
          float **weight_diff_ptrs = (float **)weight_diff_ptrs_->mutable_cpu_data();
          for (int j = 0; j < tile_h_in_*tile_w_in_*this->group_; ++j) {
            weight_diff_ptrs[j] =
              this->blobs_[0]->mutable_gpu_diff() +
              j*(this->conv_out_channels_/this->group_)*(this->conv_in_channels_/this->group_);
          }
          weight_diff_ptrs_initialized_ = true;
        }
        
        for (int n = 0; n < this->num_; ++n) {
          float alpha = 1, beta = 1;

          int M = this->conv_out_channels_/this->group_;
          int N = this->conv_in_channels_/this->group_;
          int K = ntiles_h_*ntiles_w_;

          CUBLAS_CHECK(hipblasSgemmBatched(
            Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N,
            N, M, K,
            &alpha,
            (const float **)in_activation_ptrs_->gpu_data() + n*tile_h_in_*tile_w_in_*this->group_, K,
            (const float **)out_activation_ptrs_->gpu_data() + n*tile_h_in_*tile_w_in_*this->group_, K,
            &beta,
            (float **)weight_diff_ptrs_->mutable_gpu_data(), N,
            tile_h_in_*tile_w_in_*this->group_));
            // weight_diff has (tile_h_in*tile_w_in) x (conv_out_channels) x (conv_in_channels/group) dimension
          
#if 0
          const float *weight_diff_cpu = this->blobs_[0]->cpu_diff();
          for (int i = 0; i < tile_h_in_*tile_w_in_*this->conv_out_channels_*(this->conv_in_channels_/this->group_); ++i) {
            if (isnan(weight_diff_cpu[i])) {
              ostringstream str;
              str << "nan at weight_diff[" << i << "]";
              LOG(FATAL) << str.str();
            }
          }
#endif          

          if (false/*n == this->num_ - 1*/) {
            float *temp_weight = NULL;
            size_t len = this->conv_out_channels_*(this->conv_in_channels_/this->group_)*kernel_h*kernel_w;
            CUDA_CHECK(hipMalloc(&temp_weight, sizeof(float)*len));

            caffe_gpu_gemm<float>(CblasTrans, CblasNoTrans,
                this->conv_out_channels_*(this->conv_in_channels_/this->group_), kernel_h*kernel_w, tile_h_in_*tile_w_in_,
                (float)1, weight_diff, GKronG->get()->gpu_data(),
                (float)0, temp_weight);
                
            float *temp_weight_cpu = new float[len];
            CUDA_CHECK(hipFree(temp_weight));
            CUDA_CHECK(hipMemcpy(temp_weight_cpu, temp_weight, sizeof(float)*len, hipMemcpyDeviceToHost));

            fprintf(stderr, "weight_diff[%d]\n", n);
            for (int m = 0; m < this->conv_out_channels_; ++m) {
              for (int c = 0; c < this->conv_in_channels_/this->group_; ++c) {
                for (int i = 0; i < kernel_h*kernel_w; ++i) {
                  fprintf(stderr, "%g ", temp_weight_cpu[(m*(this->conv_in_channels_/this->group_) + c)*kernel_h*kernel_w + i]);
                }
              }
              fprintf(stderr, "\n");
            }
            delete[] temp_weight_cpu;

            const float *weight_diff_cpu = this->blobs_[0]->cpu_diff();
            fprintf(stderr, "weight_diff_winograd[%d]\n", n);
            for (int n = 0; n < this->conv_out_channels_; ++n) {
              for (int c = 0; c < this->conv_in_channels_; ++c) {
                for (int j = 0; j < tile_h_in_*tile_w_in_; ++j) {
                  fprintf(stderr, "%g ", weight_diff_cpu[(j*this->conv_out_channels_ + n)*this->conv_in_channels_ + c]);
                }
              }
              fprintf(stderr, "\n");
            }
          }
        } // for each input
        
#ifdef PROFILE_WINOGRAD
        LOG(INFO) << "Convolution for weight gradient takes " << timer.MicroSeconds()/1e6;
#endif
      } // param_propagate_down_[0]

      // gradient w.r.t. bottom data, if necessary.
      if (propagate_down[i]) {
#ifdef PROFILE_WINOGRAD
        timer.Start();
#endif
        // Convolution in Winograd domain
        float alpha = 1, beta = 0;
        int M = this->conv_in_channels_/this->group_;
        int N = ntiles_h_*ntiles_w_;
        int K = this->conv_out_channels_/this->group_;

        CUBLAS_CHECK(hipblasSgemmBatched(
          Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T,
          N, M, K,
          &alpha,
          (const float **)out_activation_ptrs_->gpu_data(), N,
          (const float **)weight_ptrs_->gpu_data(), M,
          &beta,
          (float **)in_activation_ptrs_->mutable_gpu_data(), N,
          in_activation_ptrs_->count()));
#ifdef PROFILE_WINOGRAD
        LOG(INFO) << "Convolution for bottom gradient takes " << timer.MicroSeconds()/1e6;
#endif

        // Transform back to time domain
#ifdef PROFILE_WINOGRAD
        timer.Start();
#endif
        caffe_gpu_gemm<float>(CblasTrans, CblasTrans,
            this->num_*this->conv_in_channels_*ntiles_h_*ntiles_w_, tile_h_in_*tile_w_in_, tile_h_in_*tile_w_in_,
            (float)1, temp1_.mutable_gpu_data(), BKronB->get()->gpu_data(),
            (float)0, this->col_buffer_.mutable_gpu_data());
#ifdef PROFILE_WINOGRAD
        LOG(INFO) << "Inverse transformation of bottom_diff takes " << timer.MicroSeconds()/1e6;
#endif

#ifdef PROFILE_WINOGRAD
        timer.Start();
#endif
        num_kernels = this->num_*this->conv_in_channels_*ntiles_h_*ntiles_w_*tile_h_in_*tile_w_in_;

        CUDA_CHECK(hipMemset(bottom_diff, 0, sizeof(float)*this->num_*this->conv_in_channels_*height*width));
        winograd_input_col2im_gpu_kernel<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                                  CAFFE_CUDA_NUM_THREADS>>>(
          num_kernels,
          this->col_buffer_.gpu_data(), bottom_diff,
          height, width,
          pad_h, pad_w,
          ntiles_h_, ntiles_w_,
          tile_h_in_, tile_w_in_,
          tile_h_out_, tile_w_out_);
#ifdef PROFILE_WINOGRAD
        LOG(INFO) << "winograd_input_col2im takes " << timer.MicroSeconds()/1e6;
#endif

#if 0
        const float *bottom_diff_cpu = bottom[i]->cpu_diff();
        for (int i = 0; i < this->bottom_dim_; ++i) {
          if (isnan(bottom_diff_cpu[i])) {
            ostringstream str;
            str << "nan at bottom_diff[" << n << ", " << i << "]";
          }
        }
#endif
      } // propagate_down_[i]
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(WinogradLayer);

}  // namespace caffe
